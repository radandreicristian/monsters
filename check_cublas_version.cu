
#include <hip/hip_runtime.h>
#include <cstdio>
#include <hipblas.h>

int main() {
    hipblasHandle_t handle;
    // Create a cuBLAS handle
    if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    
    int version = 0;
    // Get the version of cuBLAS
    if (cublasGetVersion_v2(handle, &version) != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "Failed to get cuBLAS version\n");
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    printf("cuBLAS version: %d\n", version);
    hipblasDestroy(handle);
    return EXIT_SUCCESS;
}
